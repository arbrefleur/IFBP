#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "para.h"
// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h> // helper functions for SDK examples

float *d_gg=NULL;
__constant__ float gC_angle_sin[frameN];
__constant__ float gC_angle_cos[frameN];

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { getchar(); exit(code); }
	}
}
__global__ void gpu_fdk(float *d_gg,float *d_outatemp,int width,int height,int bottom,int k)
{
	const int i = blockDim.x*blockIdx.x + threadIdx.x;

	const int j = blockDim.y*blockIdx.y+threadIdx.y;
	//int k=blockDim.z*blockIdx.z+threadIdx.z;
	if((i<width)&&(j<height)){
		int m;

		float dlta=2*pi/frameN;
		float x=(float)(i-width/2);
		float y=(float)(j-height/2);		
		int	 z=k+bottom;
		float temp=0.0f;
			for( m =0;m<frameN;m++)
			{
				const float cos_theta = gC_angle_cos[m];
				const float sin_theta = gC_angle_sin[m];
				double tempt00 = x*sin_theta*sinf(Angle*pi/180)- y*cos_theta*sinf(Angle*pi/180)+z*cosf(Angle*pi/180)+ODD-DIS;
				float prjx= -DIS*(x*cos_theta + y*sin_theta)/tempt00 + DetectX/2+ CX;
				float prjy=-DIS*(-x*sin_theta*cosf(Angle*pi/180)+y*cos_theta*cosf(Angle*pi/180)+z*sinf(Angle*pi/180))/tempt00 + DetectZ/2+ CZ;
				int xd = (int)prjx;
				float xf = prjx -xd;
				int yd = (int) prjy;
				float yf = prjy -yd;
				if((0<xd)&&(xd<DetectX-1) &&( yd>0)&& yd<(DetectZ-1))
				{
					float szd = (1-xf)**(d_gg+m*DetectX*DetectZ+yd*DetectX+xd) + xf**(d_gg+m*DetectX*DetectZ+yd*DetectX+xd+1);
					float szd1 = (1-xf)**(d_gg+m*DetectX*DetectZ+(yd+1)*DetectX+xd) + xf**(d_gg+m*DetectX*DetectZ+(yd+1)*DetectX+xd+1);
					float z0x0y0 = (1-yf)*szd + yf*szd1;
					 temp += z0x0y0*dlta;
				}
				*(d_outatemp+k*height*width+i*width+j)=temp;//	a[i][j] = a[i][j] + Xs*dltaBeta;
			}
		
		
		//if(*(d_outatemp+k*height*width+i*width+j)<1e-6)*(d_outatemp+k*height*width+i*width+j)=0;
	}
}
__global__ void gpu_projection()
{

}
extern "C"
void gpu_fdkbackprj(float *prj,float *img,int height,int width,int imageW,int imageH,int bottom,int top)
{
	int k;
	unsigned int i;
	const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
	const dim3 gridSize((imageW+blockSize.x-1) / blockSize.x, (imageH+blockSize.y-1) / blockSize.y,1);
	float *d_outimg=NULL;
	
	gpuErrchk(hipMalloc((void**)&d_outimg,imageH*imageW*(top-bottom)*sizeof(float)));
	gpuErrchk(hipMemset((void*)d_outimg,0,imageH*imageW*(top-bottom)))
	gpuErrchk(hipMemcpy(d_gg,prj,height*width*frameN*sizeof(float),hipMemcpyHostToDevice));
	float* angle_sin = new float[frameN];
	float* angle_cos = new float[frameN];
	float* angles = new float[frameN];
	float fai_sin;float fai_cos;
	fai_sin=sinf(Angle*pi/180);
	fai_cos=cosf(Angle*pi/180);
	for ( i = 0; i < frameN; ++i)
		angles[i] = (frameN-1-i)*pi/180;
	for ( i = 0; i < frameN; ++i) {
		angle_sin[i] = sinf(angles[i]);
		angle_cos[i] = cosf(angles[i]);
	}
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_sin), angle_sin, frameN*sizeof(float), 0, hipMemcpyHostToDevice));
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_cos), angle_cos, frameN*sizeof(float), 0, hipMemcpyHostToDevice));
	
for(k=0;k<top-bottom;k++)	
	{
		
		gpu_fdk<<<gridSize, blockSize>>>(d_gg,d_outimg,imageW,imageH,bottom,k);
		gpuErrchk(hipPeekAtLastError());
		hipDeviceSynchronize();
	}
	gpuErrchk(hipMemcpy(img,d_outimg,imageW*imageH*(top-bottom)*sizeof(float),hipMemcpyDeviceToHost));
}
////////////////////////////////////////////////////////////////////////////////
extern "C"
void gpu_molloc(int width,int height,int frames)
{	
	gpuErrchk(hipMalloc((void**)&d_gg,width*height*frames*sizeof(float)));
}
extern "C"
void gpu_release()
{
	gpuErrchk(hipFree(d_gg));
}