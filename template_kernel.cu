#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "para.h"

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h> // helper functions for SDK examples
//texture<float, 3, hipReadModeElementType> gg_tex;
typedef texture<float, 3, hipReadModeElementType> texture3D;
static texture3D img_tex;
//float *d_gg=NULL;
__constant__ float gC_angle_sin[frameN];
__constant__ float gC_angle_cos[frameN];
__constant__ float gc_WDO[frameN];

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { getchar(); exit(code); }
	}
}
__device__ void sortdifferent(long* out, double* X,double* Y,double* Z,int Z0, int Z1)
{
	int i,j,k;
	i =0;j=0;
	for( k =0;i<Z0 || j<Z1;k++)
	{

		if(i<Z0 && j< Z1)
		{
			if(fabs(X[i] - Y[j])<1e-6 )
			{ 
				Z[k] = X[i++];
				j++; 
			}
			else if( X[i] < Y[j]) 
			{
				Z[k] = X[i++];
			}
			else
			{
				Z[k] = Y[j++];
			}

		}
		else{
			if(i<Z0) Z[k] = X[i++];
			if(j<Z1) Z[k] = Y[j++];

		}


	}
	*out=k;

}
__device__ void get_prjindex(float cos_theta,float sin_theta,int x,int y,int z,int* xmin,int* xmax,int* ymin,int* ymax)
{
	int i;
	float u,v,w,minxtmp,maxxtmp,minytmp,maxytmp,prjx[8],prjy[8];
	u=x*cos_theta-y*sin_theta;
	v=cosf(alpha*pi/180)*(y*cos_theta+x*sin_theta)-z*sinf(alpha*pi/180);
	w=z*cosf(alpha*pi/180)+(y*cos_theta+x*sin_theta)*sinf(alpha*pi/180);			
	float temp00=(ODD-v)/(-FOD-v);
	minxtmp=maxxtmp=prjx[0]=temp00*(-u)+u+DetectX/2+ CX;
	minytmp=maxytmp=prjy[0]=-(temp00*(-w)+w)+DetectZ/2+ CZ;
	u=(x+1)*cos_theta-y*sin_theta;
	v=cosf(alpha*pi/180)*(y*cos_theta+(x+1)*sin_theta)-z*sinf(alpha*pi/180);
	w=z*cosf(alpha*pi/180)+(y*cos_theta+(x+1)*sin_theta)*sinf(alpha*pi/180);			
 temp00=(ODD-v)/(-FOD-v);
	prjx[1]=temp00*(-u)+u+DetectX/2+ CX;
	prjy[1]=-(temp00*(-w)+w)+DetectZ/2+ CZ;
	u=x*cos_theta-(y+1)*sin_theta;
	v=cosf(alpha*pi/180)*((y+1)*cos_theta+x*sin_theta)-z*sinf(alpha*pi/180);
	w=z*cosf(alpha*pi/180)+((y+1)*cos_theta+x*sin_theta)*sinf(alpha*pi/180);			
 temp00=(ODD-v)/(-FOD-v);
	prjx[2]=temp00*(-u)+u+DetectX/2+ CX;
	prjy[2]=-(temp00*(-w)+w)+DetectZ/2+ CZ;
	u=(x+1)*cos_theta-(y+1)*sin_theta;
	v=cosf(alpha*pi/180)*((y+1)*cos_theta+(x+1)*sin_theta)-z*sinf(alpha*pi/180);
	w=z*cosf(alpha*pi/180)+((y+1)*cos_theta+(x+1)*sin_theta)*sinf(alpha*pi/180);			
 temp00=(ODD-v)/(-FOD-v);
	prjx[3]=temp00*(-u)+u+DetectX/2+ CX;
	prjy[3]=-(temp00*(-w)+w)+DetectZ/2+ CZ;
	u=(x+1)*cos_theta-(y+1)*sin_theta;
	v=cosf(alpha*pi/180)*((y+1)*cos_theta+(x+1)*sin_theta)-(z+1)*sinf(alpha*pi/180);
	w=(z+1)*cosf(alpha*pi/180)+((y+1)*cos_theta+(x+1)*sin_theta)*sinf(alpha*pi/180);			
 temp00=(ODD-v)/(-FOD-v);
	prjx[4]=temp00*(-u)+u+DetectX/2+ CX;
	prjy[4]=-(temp00*(-w)+w)+DetectZ/2+ CZ;
	u=(x)*cos_theta-(y)*sin_theta;
	v=cosf(alpha*pi/180)*((y)*cos_theta+(x)*sin_theta)-(z+1)*sinf(alpha*pi/180);
	w=(z+1)*cosf(alpha*pi/180)+((y)*cos_theta+(x)*sin_theta)*sinf(alpha*pi/180);			
 temp00=(ODD-v)/(-FOD-v);
	prjx[5]=temp00*(-u)+u+DetectX/2+ CX;
	prjy[5]=-(temp00*(-w)+w)+DetectZ/2+ CZ;
	u=(x+1)*cos_theta-(y)*sin_theta;
	v=cosf(alpha*pi/180)*((y)*cos_theta+(x+1)*sin_theta)-(z+1)*sinf(alpha*pi/180);
	w=(z+1)*cosf(alpha*pi/180)+((y)*cos_theta+(x+1)*sin_theta)*sinf(alpha*pi/180);			
 temp00=(ODD-v)/(-FOD-v);
	prjx[6]=temp00*(-u)+u+DetectX/2+ CX;
	prjy[6]=-(temp00*(-w)+w)+DetectZ/2+ CZ;
	u=(x)*cos_theta-(y+1)*sin_theta;
	v=cosf(alpha*pi/180)*((y+1)*cos_theta+(x)*sin_theta)-(z+1)*sinf(alpha*pi/180);
	w=(z+1)*cosf(alpha*pi/180)+((y+1)*cos_theta+(x)*sin_theta)*sinf(alpha*pi/180);			
 temp00=(ODD-v)/(-FOD-v);
	prjx[7]=temp00*(-u)+u+DetectX/2+ CX;
	prjy[7]=-(temp00*(-w)+w)+DetectZ/2+ CZ;
	for(i=1;i<7;i++)
	{
		if(prjx[i]<minxtmp)minxtmp=prjx[i];
		if(prjy[i]<minytmp)minytmp=prjy[i];
		if(prjx[i]>maxxtmp)maxxtmp=prjx[i];
		if(prjy[i]>maxxtmp)maxxtmp=prjy[i];
	}
	*xmin=(int)minxtmp;*xmax=(int)maxxtmp;*ymin=(int)minytmp;*ymax=(int)maxytmp;
}
__device__ void get_lenth(float cos_theta,float sin_theta,int xd,int yd,int x,int y,int z,float* len)
{
		float s1=-FOD*cosf(alpha*pi/180)*sin_theta;
		float s2=-FOD*cosf(alpha*pi/180)*cos_theta;
		float s3=FOD*sin(alpha*pi/180);
		float d1=(xd-DetectX/2-CX)*cos_theta+sin_theta*(ODD*cosf(alpha*pi/180)+(DetectZ/2-yd)*sinf(alpha*pi/180));
		float d2=-(xd-DetectX/2-CX)*sin_theta+(ODD*cosf(alpha*pi/180)+(DetectZ/2-yd)*sin(alpha*pi/180))*cos_theta;
		float d3=-ODD*sin(alpha*pi/180)+(DetectZ/2-yd)*cosf(alpha*pi/180);
		int aflag=0;int bflag=0;
		
		float xa,ya,za,xb,yb,zb,xtemp,ytemp,ztemp;
		xtemp=(z+1-s3)*(d1-s1)/(d3-s3)+s1;ytemp=(z+1-s3)*(d2-s2)/(d3-s3)+s2;
		if(xtemp<=x+1&&xtemp>=x&&ytemp>=y&&ytemp<=y+1&&(!aflag||!bflag))
		{
			xa=xtemp;ya=ytemp;za=z+1;aflag=1;
		}
		xtemp=(z-s3)*(d1-s1)/(d3-s3)+s1;ytemp=(z-s3)*(d2-s2)/(d3-s3)+s2;
		if(xtemp<=x+1&&xtemp>=x&&ytemp>=y&&ytemp<=y+1&&(!aflag||!bflag))
		{
			if(aflag){xb=xtemp;yb=ytemp;zb=z;bflag=1;}else {xa=xtemp;ya=ytemp;za=z;aflag=1;}
		}
		ytemp=(x-s1)*(d2-s2)/(d1-s1)+s2;ztemp=(x-s1)*(d3-s3)/(d1-s1)+s3;
		if(ztemp<=z+1&&ztemp>=z&&ytemp>=y&&ytemp<=y+1&&(!aflag||!bflag))
		{
			if(aflag){xb=x;yb=ytemp;zb=ztemp;bflag=1;}else {xa=x;ya=ytemp;za=ztemp;aflag=1;}
		}
		ytemp=(x+1-s1)*(d2-s2)/(d1-s1)+s2;ztemp=(x+1-s1)*(d3-s3)/(d1-s1)+s3;
		if(ztemp<=z+1&&ztemp>=z&&ytemp>=y&&ytemp<=y+1&&(!aflag||!bflag))
		{
			if(aflag){xb=x+1;yb=ytemp;zb=ztemp;bflag=1;}else {xa=x+1;ya=ytemp;za=ztemp;aflag=1;}
		}
		xtemp=(y-s2)*(d1-s1)/(d2-s2)+s1;ztemp=(y-s2)*(d3-s3)/(d2-s2)+s3;
		if(ztemp<=z+1&&ztemp>=z&&xtemp<=x+1&&xtemp>=x&&(!aflag||!bflag))
		{
			if(aflag){xb=xtemp;yb=y;zb=ztemp;bflag=1;}else {xa=xtemp;ya=y;za=ztemp;aflag=1;}
		}
		xtemp=(y+1-s2)*(d1-s1)/(d2-s2)+s1;ztemp=(y+1-s2)*(d3-s3)/(d2-s2)+s3;
		if(ztemp<=z+1&&ztemp>=z&&xtemp<=x+1&&xtemp>=x&&(!aflag||!bflag))
		{
			if(aflag){xb=xtemp;yb=y+1;zb=ztemp;bflag=1;}else {xa=xtemp;ya=y+1;za=ztemp;aflag=1;}
		}
		if(aflag&&bflag)
		{
			*len=sqrt((xa-xb)*(xa-xb)+(ya-yb)*(ya-yb)+(za-zb)*(za-zb));
		}else *len=0.0f;
}
__global__ void gpu_backProjection(float *d_gg,float *d_outatemp,const int width,const int height,int bottom,int top)
{
	const int i = blockDim.x*blockIdx.x + threadIdx.x;
	const int j = blockDim.y*blockIdx.y+threadIdx.y;
	int xmin,xmax,ymin,ymax,xd,yd;
	//int k=blockDim.z*blockIdx.z+threadIdx.z;
	//float max_r=0.0f;
	//max_r=(float)(i-width/2)*(i-width/2)/((width/2)*(width/2))+(float)(j-height/2)*(j-height/2)/((height/2)*(height/2));
	if((i<width)&&(j<height)){
	//if(max_r<1.0){
	
		int k,m,z;

		float x=(float)(i-width/2);
		float y=(float)(-j+height/2);
		float u,v,w,prjx[8],prjy[8];
		for(k=0;k<top-bottom;k++)
		{
			z=k+bottom;	
			float temp=0.0f,len=0.0f;
			for(m=0;m<2;m++)
			{
				const float cos_theta =gC_angle_cos[m];// gC_angle_cos[frameN-1-m];
				const float sin_theta =gC_angle_sin[m];// gC_angle_sin[frameN-1-m];				
				get_prjindex(cos_theta,sin_theta,x,y,z,&xmin,&xmax,&ymin,&ymax);
				 for(yd=ymin;yd<ymax;yd++)
					 for(xd=xmin;xd<xmax;xd++)
					 {
						 get_lenth(cos_theta,sin_theta,xd,yd,x,y,z,&len);
						 if(len>0)
						 {
							 temp+=len**(d_gg+m*DetectX*DetectZ+yd*DetectX+xd);
						 }
					 }
			
			}
			//if(temp<1e-6)
				//temp=0;
			*(d_outatemp+k*height*width+j*width+i)=temp;
		
		}
		//if(*(d_outatemp+k*height*width+j*width+i)<1e-6)*(d_outatemp+k*height*width+j*width+i)=0;
	}
}
__global__ void get_projection(float* img,float* prj,double* z0,double* zx,double* zy,double* zz,long* J,double* L)
{	
	const int k = blockDim.x*blockIdx.x + threadIdx.x;
	const int n = blockDim.y*blockIdx.y+threadIdx.y;
	float d1,d2;
	d1 = n -DetectZ/2;
	d2 = k - DetectX/2;


	long int i,j;
	double x1,y1,z1,x2,y2,z2;
	double xs,ys,zs,xd,yd,zd;

	/*double y= n-imageY/2;
	double x = k-imageX/2;
	x1 = FOD*sintable[m]*sinfai+imageX/2;
	y1 =-FOD*costable[m]*sinfai+imageY/2;
	z1 =FOD*cosfai-zbot;
	x2=x*costable[m]-y*sintable[m]*cosfai-ODD*sintable[m]*sinfai+imageX/2;
	y2=x*sintable[m]+y*costable[m]*cosfai+ODD*costable[m]*sinfai+imageY/2;
	z2=y*sinfai-ODD*cosfai-zbot;
	*/
	
	int m;
	if(k<DetectX&&n<DetectZ)
		for( m =0;m<frameN;m++)
		{
			const float cos_theta = gC_angle_cos[m];
			const float sin_theta = gC_angle_sin[m];
			double y= n;
			double x = k;
			x1 = -FOD*cosf(alpha*pi/180)*sin_theta+imageX/2;
			y1 =-FOD*cosf(alpha*pi/180)*cos_theta+imageY/2;
			z1 =FOD*sin(alpha*pi/180);
			x2=(x-DetectX/2-CX)*cos_theta+sin_theta*(ODD*cosf(alpha*pi/180)+(DetectZ/2-y)*sinf(alpha*pi/180))+imageX/2;
			y2=-(x-DetectX/2-CX)*sin_theta+(ODD*cosf(alpha*pi/180)+(DetectZ/2-y)*sin(alpha*pi/180))*cos_theta+imageY/2;
			z2=-ODD*sin(alpha*pi/180)+(DetectZ/2-y)*cosf(alpha*pi/180)-zbot;
		
	
	
	double kyx,kzx,kxy,kzy,kxz,kyz;
	double byx,bzx,bxy,bzy,bxz,byz;
	double zxN,zyN,yxN,yzN,xyN,xzN;
	double xmin,xmax,ymin,ymax,zmin,zmax;

	long int c1=0,c2=0,c3=0;
	for(i = 0;i<=3*imageN;i++)
	{
		z0[i] = 0;
		zx[i] = 0;
		zy[i] = 0;
		zz[i] = 0;
		J[i] = 0;
		L[i] = 0;
	}


	// x1 != x2  y1 != y2   z1 != z2 
	if( fabs(x1-x2) > 1e-6  && fabs(y1-y2)>1e-6 &&  fabs(z1-z2)>1e-6  )
	{

		kzx = (z2-z1)/(x2-x1);  bzx=z1-kzx*x1; zxN = kzx*imageX + bzx;
		kzy = (z2-z1)/(y2-y1);  bzy=z1-kzy*y1; zyN = kzy*imageY + bzy;
		kyx = (y2-y1)/(x2-x1);  byx=y1-kyx*x1; yxN = kyx*imageX + byx;
		kyz = (y2-y1)/(z2-z1);  byz=y1-kyz*z1; yzN = kyz*(ztop-zbot) + byz;
		kxy = (x2-x1)/(y2-y1);  bxy=x1-kxy*y1; xyN = kxy*imageY + bxy;
		kxz = (x2-x1)/(z2-z1);  bxz=x1-kxz*z1; xzN = kxz*(ztop-zbot) + bxz;

		if(kzx>0) { zmax = (ztop-zbot)<zxN ? (ztop-zbot):zxN; zmin = 0>bzx ? 0:bzx;	}
		else      { zmax = (ztop-zbot)<bzx ? (ztop-zbot):bzx; zmin = 0>zxN ? 0:zxN;	}


		if(zmin >= ztop-zbot || zmax <=0 )
			c1 = 0;
		else	
		{
			if(kzy>0) { zmax = zmax<zyN ? zmax:zyN;  zmin = zmin>bzy ? zmin:bzy;	}
			else      { zmax = zmax<bzy ? zmax:bzy;  zmin = zmin>zyN ? zmin:zyN;	}

			if(kyx>0) { ymax = imageY<yxN ? imageY:yxN; ymin = 0>byx ? 0:byx;	}
			else      { ymax = imageY<byx ? imageY:byx; ymin = 0>yxN ? 0:yxN;	}
			if(kyz>0) { ymax = ymax<yzN ? ymax:yzN;  ymin = ymin>byz ? ymin:byz;	}
			else      { ymax = ymax<byz ? ymax:byz;  ymin = ymin>yzN ? ymin:yzN;	}

			if(kxy>0) { xmax = imageX<xyN ? imageX:xyN; xmin = 0>bxy ? 0:bxy;	}
			else      { xmax = imageX<bxy ? imageX:bxy; xmin = 0>xyN ? 0:xyN;	}
			if(kxz>0) { xmax = xmax<xzN ? xmax:xzN;  xmin = xmin>bxz ? xmin:bxz;	}
			else      { xmax = xmax<bxz ? xmax:bxz;  xmin = xmin>xzN ? xmin:xzN;	}

			for(j=0,i=ceil(zmin);i<=(int)(zmax);i++)
			{ z0[j++] = i; 
			if(z0[j-1]>ztop-zbot) z0[j-1] = ztop-zbot;
			}
			c1 = j;

			if(kzy>0)
			{
				for(j=0,i=ceil(ymin);i<=floor(ymax);i++)
				{ zy[j++] = kzy*i + bzy;
				if(zy[j-1]>ztop-zbot) zy[j-1] = ztop-zbot; 
				}
				c2 =j;
			}
			else
			{for(j=0,i=floor(ymax);i>=ceil(ymin);i--)
			{ zy[j++] = kzy*i + bzy; 
			if(zy[j-1]>ztop-zbot) zy[j-1] = ztop-zbot;
			if(zy[j-1]<0) zy[j-1] = 0;
			}
			c2 =j;
			}

			if(kzx>0)
			{for(j=0,i=ceil(xmin);i<=floor(xmax);i++)
			{ zx[j++] = kzx*i + bzx; 
			if(zx[j-1]>ztop-zbot) zx[j-1] = ztop-zbot;
			if(zx[j-1]<0) zx[j-1] = 0;
			}
			c3 =j;
			}
			else
			{for(j=0,i=floor(xmax);i>=ceil(xmin);i--)
			{ zx[j++] = kzx*i + bzx; 
			if(zx[j-1]>ztop-zbot) zx[j-1] = ztop-zbot;
			if(zx[j-1]<0) zx[j-1] = 0;
			}
			c3 =j;
			}

			//	for(i=0;i<c2;i++)
			//		printf("与y轴共有个%d交点，交点zy[%d]=%.6f\n",c2,i,zy[i]);
			//		for(i=0;i<c3;i++)
			// 		printf("与x轴共有个%d交点，交点zx[%d]=%.6f\n",c3,i,zx[i]);
			sortdifferent(&c1,z0,zy,zz,c1,c2);
			sortdifferent(&c1,zz,zx,z0,c1,c3);
			//	for(i=0;i<c1;i++)
			//	printf("与物体共有个%d交点，交点z0[%d]=%.6f\n",c1,i,z0[i]);
			for(i=0;i<c1-1;i++)
			{
				J[i] = (int)(kxz*(z0[i+1]+z0[i])/2 + bxz) + ((int)(kyz*(z0[i+1]+z0[i])/2 + byz))*imageX +((int)((z0[i+1]+z0[i])/2))*imageX*imageY;
				L[i] = fabs(z0[i+1]-z0[i])*sqrt(1+kxz*kxz +kyz*kyz);
			}
			float bmt;
			bmt = 0;
		}
	}

	// x1 = x2 ,y1 !=y2, z1 !=z2
	else if( fabs(x1-x2) <= 1e-6  && fabs(y1-y2)>1e-6 &&  fabs(z1-z2)>1e-6  )
	{
		if(x1 >0 &&x1 <imageX)
		{

			kzy = (z2-z1)/(y2-y1);  bzy=z1-kzy*y1; zyN = kzy*imageY + bzy;
			kyz = (y2-y1)/(z2-z1);  byz=y1-kyz*z1; yzN = kyz*(ztop-zbot) + byz;

			if(kzy>0) { zmax = (ztop-zbot)<zyN ? (ztop-zbot):zyN;  zmin = 0>bzy ? 0:bzy;	}
			else      { zmax = (ztop-zbot)<bzy ? (ztop-zbot):bzy;  zmin =0>zyN ? 0:zyN;	}
			if(zmin >= ztop-zbot || zmax <=0 )
				c1 = 0;
			else{
				if(kyz>0) { ymax = imageY<yzN ? imageY:yzN;  ymin = 0>byz ? 0:byz;	}
				else      { ymax = imageY<byz ? imageY:byz;  ymin = 0>yzN ? 0:yzN;	}

				for(j=0,i=ceil(zmin);i<=floor(zmax);i++)
				{ z0[j++] = i; 
				if(z0[j-1]>ztop-zbot) z0[j-1] = ztop-zbot;
				}
				c1 = j;

				if(kyz>0)
				{
					for(j=0,i=ceil(ymin);i<=floor(ymax);i++)
					{ zy[j++] = kzy*i + bzy;
					if(zy[j-1]>ztop-zbot) zy[j-1] = ztop-zbot; 
					}
					c2 =j;
				}
				else
				{for(j=0,i=floor(ymax);i>=ceil(ymin);i--)
				{ zy[j++] = kzy*i + bzy; 
				if(zy[j-1]>ztop-zbot) zy[j-1] = ztop-zbot;
				}
				c2 =j;
				}


			    sortdifferent(&c1,z0,zy,zz,c1,c2);
				for(i=0;i<c1;i++)
					z0[i] = zz[i];
				for(i=0;i<c1-1;i++)
				{
					J[i] = (int)((x1+x2)/2) + ((int)(kyz*(z0[i+1]+z0[i])/2 + byz))*imageX +((int)((z0[i+1]+z0[i])/2))*imageX*imageY;
					L[i] = fabs(z0[i+1]-z0[i])*sqrt(1 +kyz*kyz);
				}
			}

		}	else
			c1 =0;

	}


	// x1 != x2,y1 = y2,z1 !=z2 
	else if( fabs(x1-x2) > 1e-6  && fabs(y1-y2)<=1e-6 &&  fabs(z1-z2)>1e-6  )
	{
		if(y1>0 && y1<imageY){

			kzx = (z2-z1)/(x2-x1);  bzx=z1-kzx*x1; zxN = kzx*imageX + bzx;
			kxz = (x2-x1)/(z2-z1);  bxz=x1-kxz*z1; xzN = kxz*(ztop-zbot) + bxz;

			if(kzx>0) { zmax = (ztop-zbot)<zxN ? (ztop-zbot):zxN; zmin = 0>bzx ? 0:bzx;	}
			else      { zmax = (ztop-zbot)<bzx ? (ztop-zbot):bzx; zmin = 0>zxN ? 0:zxN;	}
			if(zmin >= ztop-zbot || zmax <=0 )
				c1 = 0;
			else
			{

				if(kxz>0) { xmax = imageX<xzN ? imageX:xzN;  xmin = 0>bxz ? 0:bxz;	}
				else      { xmax = imageX<bxz ? imageX:bxz;  xmin = 0>xzN ? 0:xzN;	}

				for(j=0,i=ceil(zmin);i<=floor(zmax);i++)
				{ z0[j++] = i; 
				if(z0[j-1]>ztop-zbot) z0[j-1] = ztop-zbot;
				}  
				c1 = j;


				if(kxz>0)
				{for(j=0,i=ceil(xmin);i<=floor(xmax);i++)
				{ zx[j++] = kzx*i + bzx; 
				if(zx[j-1]>ztop-zbot) zx[j-1] = ztop-zbot;
				}
				c3 =j;
				}
				else
				{for(j=0,i=floor(xmax);i>=ceil(xmin);i--)
				{ zx[j++] = kzx*i + bzx; 
				if(zx[j-1]>ztop-zbot) zx[j-1] = ztop-zbot;
				}
				c3 =j;
				}

				sortdifferent(&c1,z0,zx,zz,c1,c3);
				for(i=0;i<c1;i++)
					z0[i] = zz[i];

				for(i=0;i<c1-1;i++)
				{
					J[i] = (int)(kxz*(z0[i+1]+z0[i])/2 + bxz) + ((int)((y1+y2)/2))*imageX +((int)((z0[i+1]+z0[i])/2))*imageX*imageY;
					L[i] = fabs(z0[i+1]-z0[i])*sqrt(1+kxz*kxz);
				}
			}
		}
		else
			c1 = 0;
	}


	// x1 = x2  ,y1 = y2  z1 != z2
	else if( fabs(x1-x2) <= 1e-6  && fabs(y1-y2)<=1e-6 &&  fabs(z1-z2)>1e-6  )
	{
		if(x1 >0 && x1 <imageX && y1 >0 && y1 <imageY)
		{
			for(j=0,i =0;i<(ztop-zbot);i++) z0[j++] = i; c1 = j;
			for(i=0;i<c1-1;i++)
			{ J[i] = (int)((x1+x2)/2) + ((int)((y1+y2)/2))*imageX +((int)((z0[i+1]+z0[i])/2))*imageX*imageY;
			L[i] = 1;
			}}
		else
			c1 =0;

	}



	// x1 != x2, y1 != y2,z1 = z2
	else if( fabs(x1-x2) >1e-6  && fabs(y1-y2)>1e-6 &&  fabs(z1-z2)<=1e-6  )
	{
		if(z1 >0 && z1 <ztop-zbot){

			kyx = (y2-y1)/(x2-x1);  byx=y1-kyx*x1; yxN = kyx*imageX + byx;
			kxy = (x2-x1)/(y2-y1);  bxy=x1-kxy*y1; xyN = kxy*imageY + bxy;




			if(kyx>0) { ymax = imageY<yxN ? imageY:yxN; ymin = 0>byx ? 0:byx;	}
			else      { ymax = imageY<byx ? imageY:byx; ymin = 0>yxN ? 0:yxN;	}


			if(kxy>0) { xmax = imageX<xyN ? imageX:xyN; xmin = 0>bxy ? 0:bxy;	}
			else      { xmax = imageX<bxy ? imageX:bxy; xmin = 0>xyN ? 0:xyN;	}


			for(j=0,i=ceil(xmin);i<=floor(xmax);i++)
			{ zx[j++] = i; 
			//if(zx[j-1]>imageN) zx[j-1] = imageN;
			}
			c1 = j;

			if(kyx>0)
			{for(j=0,i=ceil(ymin);i<=floor(ymax);i++)
			{ zy[j++] = kxy*i + bxy; 
			//if(zy[j-1]>imageN) zy[j-1] = imageN;
			}
			c3 =j;
			}
			else
			{for(j=0,i=floor(ymax);i>=ceil(ymin);i--)
			{ zy[j++] = kxy*i + bxy; 
			//if(zy[j-1]>imageN) zy[j-1] = imageN;
			}
			c3 =j;
			}

			sortdifferent(&c1,zx,zy,z0,c1,c3);

			for(i=0;i<c1-1;i++)
			{
				J[i] = (int)((z0[i+1]+z0[i])/2 ) + ((int)(kyx*(z0[i+1]+z0[i])/2 + byx))*imageX +((int)((z1+z2)/2))*imageX*imageY;
				L[i] = fabs(z0[i+1]-z0[i])*sqrt(1+kyx*kyx);
			}
		}else
			c1 = 0;

	}


	//x1 = x2,y1! = y2,z1 = z2
	else if( fabs(x1-x2) <= 1e-6  && fabs(y1-y2)>1e-6 &&  fabs(z1-z2) <= 1e-6  )
	{	if(x1>0 && x1 <imageX && z1 >0 && z1 <ztop -zbot)
	{
		for(j=0,i =0;i<imageY;i++) z0[j++] = i; c1 =j;
		for(i=0;i<c1-1;i++)
		{ J[i] = (int)((x1+x2)/2) + ((int)((z0[i+1]+z0[i])/2))*imageX +((int)((z1+z2)/2))*imageX*imageY;
		L[i] = 1;
		}
	}
	else
		c1 = 0;
	}

	//x1 ! =x2 ,y1 =y2 ,z1 =z2;
	else if( fabs(x1-x2) > 1e-6 && fabs(y1-y2)<=1e-6 &&  fabs(z1-z2)<=1e-6  )
	{

		if(y1>0 && y1 <imageY && z1 >0 && z1 <ztop -zbot)
		{
			for(j=0,i =0;i<imageX;i++) z0[j++] = i; c1 =j;
			for(i=0;i<c1-1;i++)
			{ J[i] = (int)((z0[i+1]+z0[i])/2) + ((int)((y1+y2)/2))*imageY +((int)((z1+z2)/2))*imageX*imageY;
			L[i] = 1;
			}
		}
		else
			c1 =0;


	}




	if(c1 >1)
	{		
		double tempt1 =0,tempt2=0,tempt3=0;

		for( i =0;i<c1-1;i++)
		{ 
			tempt1 += L[i]*img[J[i]];
			tempt2 += L[i]*L[i];
		}


		if(tempt2<1e-6) 
		{//printf("tempt2 =%.6f\n",tempt2);
			//float ceshi;
			// scanf("%f",&ceshi);
		}else
		{
			//tempt3 = (lmta)*(gg[m][n][k] - tempt1)/tempt2;
		//	tempt3=*(prj1+m*DetectX*DetectZ+n*DetectX+k) - tempt1;
 			//if(tempt3>0)
		//		*(prj2+m*DetectX*DetectZ+n*DetectX+k) =tempt3;
//			for( i =0;i<c1-1;i++)
// 				a[J[i]] = a[J[i]]+tempt3*L[i];
				prj[m*DetectX*DetectZ+n*DetectX+k]=tempt1;
		}

	}
	}



}
__global__ void gpu_fdk(float *d_gg,float *d_outatemp,const int width,const int height,int bottom,int top)
{
	const int i = blockDim.x*blockIdx.x + threadIdx.x;

	const int j = blockDim.y*blockIdx.y+threadIdx.y;
	//int k=blockDim.z*blockIdx.z+threadIdx.z;
	float max_r=0.0f;
	max_r=(float)(i-width/2)*(i-width/2)/((width/2)*(width/2))+(float)(j-height/2)*(j-height/2)/((height/2)*(height/2));
	if((i<width)&&(j<height)){
	//if(max_r<1.0){
		int m;
		int k;
		int z;
		float dlta=2*pi/frameN;
		float x=(float)(i-width/2)-error_DX;
		float y=(float)(-j+height/2)-error_DY;
		for(k=0;k<top-bottom;k++)
		{
			z=k+bottom;
			float temp=0.0f;
			
			for( m =0;m<frameN;m++)
			{
				const float cos_theta =gC_angle_cos[m];// gC_angle_cos[frameN-1-m];
				const float sin_theta =gC_angle_sin[m];// gC_angle_sin[frameN-1-m];
				float u=x*cos_theta-y*sin_theta;
				float v=cosf(alpha*pi/180)*(y*cos_theta+x*sin_theta)-z*sinf(alpha*pi/180);
				float w=z*cosf(alpha*pi/180)+(y*cos_theta+x*sin_theta)*sinf(alpha*pi/180);
				float U2=(FOD+w)*(FOD+w)/(DIST0*DIST0);
				//float tempt00 = x*sin_theta*sinf(Angle*pi/180)- y*cos_theta*sinf(Angle*pi/180)+z*cosf(Angle*pi/180)+ODD-DIS;
				//float prjx= -DIS*(x*cos_theta + y*sin_theta)/tempt00 + DetectX/2+ CX;
				//float prjy=-DIS*(-x*sin_theta*cosf(Angle*pi/180)+y*cos_theta*cosf(Angle*pi/180)+z*sinf(Angle*pi/180))/tempt00 + DetectZ/2+ CZ;
				float temp00=(ODD-v)/(-FOD-v);
				float prjx=temp00*(-u)+u+DetectX/2+ CX;
				float prjy=-(temp00*(-w)+w)+DetectZ/2+ CZ;

				int xd = (int)prjx;
				float xf = prjx -xd;
				int yd = (int) prjy;
				float yf = prjy -yd;
				if((0<xd)&&(xd<DetectX-1) &&( yd>0)&& yd<(DetectZ-1))
				{
					float szd = (1-xf)**(d_gg+m*DetectX*DetectZ+yd*DetectX+xd) + xf**(d_gg+m*DetectX*DetectZ+yd*DetectX+xd+1);
					float szd1 = (1-xf)**(d_gg+m*DetectX*DetectZ+(yd+1)*DetectX+xd) + xf**(d_gg+m*DetectX*DetectZ+(yd+1)*DetectX+xd+1);
					float z0x0y0 = (1-yf)*szd + yf*szd1;
					temp +=z0x0y0/U2;
					// temp +=dlta* tex3D(gg_tex,prjx,prjy,m);
				}
				//	a[i][j] = a[i][j] + Xs*dltaBeta;
			}
			//if(temp<1e-6)
				//temp=0;
			*(d_outatemp+k*height*width+j*width+i)=dlta*temp;
		
		}
		//if(*(d_outatemp+k*height*width+j*width+i)<1e-6)*(d_outatemp+k*height*width+j*width+i)=0;
	}
}
__global__ void gpu_projection(float* d_output,int xmin,int xmax,int ymin,int ymax)
{
	const int idx = blockDim.x*blockIdx.x + threadIdx.x;
	const int idy = blockDim.y*blockIdx.y+threadIdx.y;
	int m;
	if(idx<DetectX&&idy<DetectZ)
		for( m =0;m<frameN;m++)
	{
		const float cos_theta = gC_angle_cos[m];
		const float sin_theta = gC_angle_sin[m];
		float s1=-FOD*cosf(alpha*pi/180)*sin_theta+error_SX;
		float s2=-FOD*cosf(alpha*pi/180)*cos_theta+error_SY;
		float s3=FOD*sin(alpha*pi/180);
		float d1=(idx-DetectX/2-CX)*cos_theta+sin_theta*(ODD*cosf(alpha*pi/180)+(DetectZ/2-idy)*sinf(alpha*pi/180))+error_DX;
		float d2=-(idx-DetectX/2-CX)*sin_theta+(ODD*cosf(alpha*pi/180)+(DetectZ/2-idy)*sin(alpha*pi/180))*cos_theta+error_DX;
		float d3=-ODD*sin(alpha*pi/180)+(DetectZ/2-idy)*cosf(alpha*pi/180);
		int aflag=0;int bflag=0;
	//x1 = SX*sintable[m]-FOD*cosf(fai*pi/180)*sintable[m]+SZ*sinf(fai*pi/180)*sintable[m]+imageX/2;
	//y1 =-SX*sintable[m]-FOD*cosf(fai*pi/180)*costable[m]+SZ*costable[m]*sinf(fai*pi/180)+imageY/2;
	//z1 =FOD*sin(fai*pi/180)+SZ*cosf(fai*pi/180)-zbot;///////-zbot
	//x2=(x-DetectX/2-CX)*costable[m]+sintable[m]*(ODD*cosf(fai*pi/180)+(DetectZ/2-y)*sinf(fai*pi/180))+imageX/2;
	//y2=-(x-DetectX/2-CX)*sintable[m]+(ODD*cosf(fai*pi/180)+(DetectZ/2-y)*sin(fai*pi/180))*costable[m]+imageY/2;
	//z2=-ODD*sin(fai*pi/180)+(DetectZ/2-y)*cosf(fai*pi/180)-zbot;
		float xa,ya,za,xb,yb,zb,xtemp,ytemp,ztemp;
		xtemp=(ztop-1-s3)*(d1-s1)/(d3-s3)+s1;ytemp=(ztop-1-s3)*(d2-s2)/(d3-s3)+s2;
		if(xtemp<=xmax&&xtemp>=xmin&&ytemp>=ymin&&ytemp<=ymax&&(!aflag||!bflag))
		{
			xa=xtemp;ya=ytemp;za=ztop-1;aflag=1;
		}
		xtemp=(zbot-s3)*(d1-s1)/(d3-s3)+s1;ytemp=(zbot-s3)*(d2-s2)/(d3-s3)+s2;
		if(xtemp<=xmax&&xtemp>=xmin&&ytemp>=ymin&&ytemp<=ymax&&(!aflag||!bflag))
		{
			if(aflag){xb=xtemp;yb=ytemp;zb=zbot;bflag=1;}else {xa=xtemp;ya=ytemp;za=zbot;aflag=1;}
		}
		ytemp=(xmin-s1)*(d2-s2)/(d1-s1)+s2;ztemp=(xmin-s1)*(d3-s3)/(d1-s1)+s3;
		if(ztemp<=ztop&&ztemp>=zbot&&ytemp>=ymin&&ytemp<=ymax&&(!aflag||!bflag))
		{
			if(aflag){xb=xmin;yb=ytemp;zb=ztemp;bflag=1;}else {xa=xmin;ya=ytemp;za=ztemp;aflag=1;}
		}
		ytemp=(xmax-s1)*(d2-s2)/(d1-s1)+s2;ztemp=(xmax-s1)*(d3-s3)/(d1-s1)+s3;
		if(ztemp<=ztop&&ztemp>=zbot&&ytemp>=ymin&&ytemp<=ymax&&(!aflag||!bflag))
		{
			if(aflag){xb=xmax;yb=ytemp;zb=ztemp;bflag=1;}else {xa=xmax;ya=ytemp;za=ztemp;aflag=1;}
		}
		xtemp=(ymin-s2)*(d1-s1)/(d2-s2)+s1;ztemp=(ymin-s2)*(d3-s3)/(d2-s2)+s3;
		if(ztemp<=ztop&&ztemp>=zbot&&xtemp<=xmax&&xtemp>=xmin<ymax&&(!aflag||!bflag))
		{
			if(aflag){xb=xtemp;yb=ymin;zb=ztemp;bflag=1;}else {xa=xtemp;ya=ymin;za=ztemp;aflag=1;}
		}
		xtemp=(ymax-s2)*(d1-s1)/(d2-s2)+s1;ztemp=(ymax-s2)*(d3-s3)/(d2-s2)+s3;
		if(ztemp<=ztop&&ztemp>=zbot&&xtemp<=xmax&&xtemp>=xmin&&(!aflag||!bflag))
		{
			if(aflag){xb=xtemp;yb=ymax;zb=ztemp;bflag=1;}else {xa=xtemp;ya=ymax;za=ztemp;aflag=1;}
		}
		if(aflag&&bflag)
		{
			float len=sqrt((xa-xb)*(xa-xb)+(ya-yb)*(ya-yb)+(za-zb)*(za-zb));
			float step=1;//步长
			float temp,x,y,z;
			float accumprj=0;
			float tx=fabs(xa-xb),ty=fabs(ya-yb),tz=fabs(za-zb);
			if(tx>=ty&&tx>=tz)
			{
				if(xa>xb){ temp=xa;xa=xb;xb=temp;temp=ya;ya=yb;yb=temp;temp=za;za=zb;zb=temp;}
				x=xa, y=ya, z=za;
				while(x<xb)
				//while((x-xa)*(xb-x)>=0)
				{
					accumprj +=tex3D(img_tex,x+imageX/2,-y+imageY/2,z-zbot);
					x+=(xb-xa)*step/len;y+=(yb-ya)*step/len;z+=(zb-za)*step/len;														
				}
			}
			else if(ty>=tx&&ty>=tz)
			{
				if(ya>yb){ temp=xa;xa=xb;xb=temp;temp=ya;ya=yb;yb=temp;temp=za;za=zb;zb=temp;}
				x=xa, y=ya, z=za;
				while(y<yb)
				//while((y-ya)*(yb-y)>=0)
				{
					accumprj +=tex3D(img_tex,x+imageX/2,-y+imageY/2,z-zbot);
					x+=(xb-xa)*step/len;y+=(yb-ya)*step/len;z+=(zb-za)*step/len;														
				}
			}
			else if(tz>=tx&&tz>=ty)
			{
				//if(za>zb){ temp=xa;xa=xb;xb=temp;temp=ya;ya=yb;yb=temp;temp=za;za=zb;zb=temp;}
				x=xa, y=ya, z=za;
				while(z<zb)
				//while((z-za)*(zb-z)>=0)
				{
					accumprj +=tex3D(img_tex,x+imageX/2,-y+imageY/2,z-zbot);
					x+=(xb-xa)*step/len;y+=(yb-ya)*step/len;z+=(zb-za)*step/len;														
				}
			}								
			if(accumprj!=0)
				d_output[DetectZ*DetectX*m+DetectX*idy+idx]=accumprj;
		}
	}
}
__global__ void gpu_Add(const float *in1,const float* in2,float* out, float scale,int size)
{
	const int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<size)
	{
		out[i]=in1[i]+scale*in2[i];
	}
}
__global__ void gpu_Getlamta(const float *in1,const float* in2,float* sum1,float* sum2,int size)
{
	extern __shared__ float shared[];
	
	const int tid= threadIdx.x;
	const int bid=blockIdx.x;
	int i;
	shared[tid]=0;
	shared[tid+MAX_THREADNUM_PERBLOCK]=0;
	for(i=bid*MAX_THREADNUM_PERBLOCK+tid;i<size;i+=BLOCKNUM*MAX_THREADNUM_PERBLOCK)
	{
		shared[tid] +=in1[i]*in1[i];
		shared[tid+MAX_THREADNUM_PERBLOCK] +=in1[i]*in2[i];
	}
	__syncthreads();
	int offset=1,mask=1;
	while(offset<MAX_THREADNUM_PERBLOCK)
	{
		if((tid&mask)==0)
		{
			shared[tid]+=shared[tid+offset];
			shared[tid+MAX_THREADNUM_PERBLOCK]+=shared[tid+offset+MAX_THREADNUM_PERBLOCK];
		}
		offset +=offset;
		mask +=offset;
		__syncthreads();
	}
	if(tid==0){
		sum1[bid]=shared[0];
		sum2[bid]=shared[MAX_THREADNUM_PERBLOCK];
	}
}
__global__ void gpu_filter(const float* hs,float* prj,int width)
{
	const int frame = blockDim.x*blockIdx.x + threadIdx.x;
	const int idy = blockDim.y*blockIdx.y+threadIdx.y;
	float ps[3*DetectX];
	int i,j;
	if(idy<DetectZ&&frame<frameN)
	{
		float tempt0,tempt1;
		float jiaquan=0.0f;
		jiaquan=DIS/sqrtf(DIS*DIS+(idy-DetectZ/2)*(idy-DetectZ/2)+(DetectX/2)*(DetectX/2));
			tempt0 = *(prj+frame*DetectZ*DetectX+idy*DetectX)*jiaquan;
			tempt1 = (*(prj+frame*DetectZ*DetectX+idy*DetectX+DetectX-1) + *(prj+frame*DetectZ*DetectX+idy*DetectX+DetectX-2))*jiaquan/2;
			//投影数据进行扩充
			for(i = 1;i<DetectX;i++) ps[i] = tempt0;
			for(i = DetectX;i<DetectX*2;i++){
				jiaquan=DIS/sqrtf(DIS*DIS+(idy-DetectZ/2)*(idy-DetectZ/2)+(i-3*DetectX/2)*(i-3*DetectX/2));
				ps[i] = *(prj+frame*DetectZ*DetectX+idy*DetectX+i-DetectX)*jiaquan;}
			for( i =2*DetectX;i<DetectX*3;i++) ps[i] =tempt1;
			//卷积滤波
			float sum ;
			for( i =0;i<DetectX;i++)
			{ 
				sum = 0;
				for( j=1;j<2*DetectX;j++ )
					sum += hs[j]*ps[i+j] ;
				*(prj+frame*DetectZ*DetectX+idy*DetectX+i) = sum/2;
			}
		
	}
}
extern "C"
void gpu_imgfilter(const float* prj,float* out,char  type[10])
{

	clock_t start,end;
	float time;
	start=clock();
	float *h,*hs;
	long i,m,l,j;
	h = new float[DetectX+1];
	hs = new float[DetectX*2];
	if(0==strcmp(type,"FH"))//FH
	{
		for( i =1;i<=DetectX;i++) 
			if(i%2==0)
				h[i]=i*cos(alpha*pi/180)/(4*pi*pi*(i*i-1));
			else
				h[i]=cos(alpha*pi/180)/(4*pi*pi*i);		
		for(i=1;i<2*DetectX;i++)
			hs[i]=h[abs(i-DetectX)+1];
	}
	
	else if(0==strcmp(type,"FL"))
	{
		
		for( i =1;i<=DetectX;i++) 
			if(i%2==0)
				h[i]=-1/(8*pi*pi*(i));
			else
				h[i]=1/(8*pi*pi*i);		
		for(i=1;i<2*DetectX;i++)
			hs[i]=h[abs(i-DetectX)+1];
	}else if(0==strcmp(type,"SL")) //if(strcmp(type,'SL'))//SL
	{
		for( i =0;i<DetectX;i++) 
			h[i] = -cos(alpha*pi/180)/(pi*pi*(4*i*i - 1));
	 	for( i =1;i<2*DetectX;i++)
	 		hs[i] = h[abs(i-DetectX)];
	}
	else if(0==strcmp(type,"RL-HN"))//RL-HN
	{
		for( i =1;i<DetectX;i++) 
			if(i%2==0)
				h[i]=(-1)*cos(alpha*pi/180)*(1/(4*pi*pi*(i+1)*(i+1))+1/(4*pi*pi*(i-1)*(i-1)));
			else
				h[i]=(-1)*cos(alpha*pi/180)/(4*pi*pi*i*i);		
		for(i=1;i<DetectX;i++)
			hs[i]=h[DetectX-i];
		for(i=DetectX+1;i<2*DetectX;i++)
			hs[i]=h[i-DetectX];
		hs[DetectX]=(1)*cos(alpha*pi/180)/(2*pi*pi);
	}
	else if(0==strcmp(type,"RL"))//RL
	{
		for( i =1;i<DetectX;i++) 
			if(i%2==0)
				h[i]=0;//(-1)*cos(alpha*pi/180)*(1/(4*pi*pi*(i+1)*(i+1))+1/(4*pi*pi*(i-1)*(i-1)));
			else
				h[i]=(-1)*cos(alpha*pi/180)/(pi*pi*i*i);		
		for(i=1;i<DetectX;i++)
			hs[i]=h[DetectX-i];
		for(i=DetectX+1;i<2*DetectX;i++)
			hs[i]=h[i-DetectX];
		hs[DetectX]=cos(alpha*pi/180)/4;
	}
	float* d_hs=NULL;
	float* d_prj=NULL;
	const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
	const dim3 gridSize((frameN+blockSize.x-1) / blockSize.x, (DetectZ+blockSize.y-1) / blockSize.y,1);
	gpuErrchk(hipMalloc((void**)&d_hs,DetectX*2*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_prj,DetectX*DetectZ*frameN*sizeof(float)));
	gpuErrchk(hipMemcpy(d_prj,prj,DetectX*DetectZ*frameN*sizeof(float),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_hs,hs,DetectX*2*sizeof(float),hipMemcpyHostToDevice));
	gpu_filter<<<gridSize,blockSize>>>(d_hs,d_prj,DetectX);
	gpuErrchk(hipMemcpy(out,d_prj,DetectZ*DetectX*frameN*sizeof(float),hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_hs));
	gpuErrchk(hipFree(d_prj));

	end=clock();
	time=(end-start)/CLK_TCK;
	printf("filter success!time=%.5f\n",time);
}
extern "C"
void theta_init()
{	
	float* angle_sin = new float[frameN];
	float* angle_cos = new float[frameN];
	float* angles = new float[frameN];
//	float fai_sin;float fai_cos;
//	fai_sin=sinf(Angle*pi/180);
//	fai_cos=cosf(Angle*pi/180);
	int i;
	for ( i = 0; i < frameN; ++i)
		if(z_dir)
			angles[i] =i*pi/180;
		else 
			angles[i] = (frameN-1-i)*pi/180;

	for ( i = 0; i < frameN; ++i) {
		angle_sin[i] = sinf(angles[i]);
		angle_cos[i] = cosf(angles[i]);
	}
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_sin), angle_sin, frameN*sizeof(float), 0, hipMemcpyHostToDevice));
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_cos), angle_cos, frameN*sizeof(float), 0, hipMemcpyHostToDevice));
	//gpuErrchk(hipMalloc((void**)&d_gg,width*height*frames*sizeof(float)));
}
extern "C"
void gpu_imgAdd(float *in1,float* in2,float* out, float scale,int size)
{
	printf("Start Adding Image...\n");
	clock_t start,end;
	float time;
	start=clock();
	float* d_output=NULL;
	float* d_in1=NULL;
	float* d_in2=NULL;
	const dim3 blockSize(MAX_THREADNUM_PERBLOCK, 1,1);
	const dim3 gridSize((size+blockSize.x-1) / blockSize.x,1, 1);
	gpuErrchk(hipMalloc((void**)&d_in1,size*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_in2,size*sizeof(float)));
	gpuErrchk(hipMemcpy(d_in1,in1,size*sizeof(float),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_in2,in2,size*sizeof(float),hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&d_output,size*sizeof(float)));
	gpu_Add<<<gridSize,blockSize>>>(d_in1,d_in2,d_output,scale,size);
	gpuErrchk(hipMemcpy(out,d_output,size*sizeof(float),hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_in1));
	gpuErrchk(hipFree(d_in2));
	gpuErrchk(hipFree(d_output));
	end=clock();
	time=(end-start)/CLK_TCK;
	printf("Image Add Succeed!time=%.5f,size=%d\n",time,size);
}
extern "C"
void gpu_getlamtak(float* out,float *in1,float* in2,int size)
{
	
	clock_t start,end;
	start=clock();
	float* d_sum1=NULL;
	float time;
	float* d_sum2=NULL;
	float* d_in1=NULL;
	float* d_in2=NULL;
	float h_sum1[BLOCKNUM]={0};
	float h_sum2[BLOCKNUM]={0};
	const dim3 gridSize(BLOCKNUM, 1,1);
	const dim3 blockSize(MAX_THREADNUM_PERBLOCK, 1,1);
//	h_sum1=(float*)malloc(BLOCKNUM*sizeof(float));
//	h_sum2=(float*)malloc(BLOCKNUM*sizeof(float));
	gpuErrchk(hipMalloc((void**)&d_in1,size*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_in2,size*sizeof(float)));
	gpuErrchk(hipMemcpy(d_in1,in1,size*sizeof(float),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_in2,in2,size*sizeof(float),hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&d_sum1,BLOCKNUM*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_sum2,BLOCKNUM*sizeof(float)));
	gpu_Getlamta<<<gridSize,blockSize,2*MAX_THREADNUM_PERBLOCK*sizeof(float)>>>(d_in1,d_in2,d_sum1,d_sum2,size);
	gpuErrchk(hipMemcpy(h_sum1,d_sum1,BLOCKNUM*sizeof(float),hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_sum2,d_sum2,BLOCKNUM*sizeof(float),hipMemcpyDeviceToHost));
	long int i;
	float final_sum1=0;
	float final_sum2=0;
	for(i=0;i<BLOCKNUM;i++)
	{
		final_sum1 +=h_sum1[i];
		final_sum2 +=h_sum2[i];
	}
	if(0!=final_sum2)
		*out=final_sum1/final_sum2;
	gpuErrchk(hipFree(d_in1));
	gpuErrchk(hipFree(d_sum1));
	gpuErrchk(hipFree(d_in2));
	gpuErrchk(hipFree(d_sum2));
	end=clock();
	time=(end-start)/CLK_TCK;
	printf("Get lamtak Succeed!lamtak=%.5f\n",*out);
}
extern "C"
void gpu_MyArtprojection( float *img,float* outgg,int xmin,int xmax,int ymin,int ymax,int depth)
{
	printf("Start Projection calculating...\n");
	clock_t start,end;
	float time;
	start=clock();
	float* d_output=NULL;
	float* d_imgin=NULL;
	//caculate_WD();
	theta_init();
	gpuErrchk(hipMalloc((void**)&d_output,DetectX*DetectZ*frameN*sizeof(float)));
	gpuErrchk(hipMemset((void*)d_output,0,DetectX*DetectZ*frameN));
	hipArray* cuArray_a;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipExtent extentA;
	extentA.width = imageX;
	extentA.height = imageY;
	extentA.depth = depth;
	gpuErrchk(hipMalloc3DArray(&cuArray_a, &channelDesc, extentA)) ;
	hipMemcpy3DParms myparms = {0};
    myparms.srcPos = make_hipPos(0,0,0);
    myparms.dstPos = make_hipPos(0,0,0);
    myparms.srcPtr = make_hipPitchedPtr(img, imageX * sizeof(float), imageX, imageX);
    myparms.dstArray = cuArray_a;
    myparms.extent = make_hipExtent(imageX, imageY, depth);
    myparms.kind = hipMemcpyHostToDevice;
    gpuErrchk(hipMemcpy3D(&myparms));

	img_tex.addressMode[0] = hipAddressModeBorder;
	img_tex.addressMode[1] = hipAddressModeBorder;
	img_tex.addressMode[2] = hipAddressModeBorder;
	img_tex.filterMode = hipFilterModeLinear;
	img_tex.normalized = false;
	hipBindTextureToArray(img_tex, cuArray_a, channelDesc);
	//gpuErrchk(hipMalloc((void**)&d_ggin,imageH*imageW*(ztop-zbot)*sizeof(float)));
	//gpuErrchk(hipMemcpy(d_ggin,img,height*width*frameN*sizeof(float),hipMemcpyHostToDevice));
	gpuErrchk(hipMemset((void*)d_output,0,DetectX*DetectZ*frameN*sizeof(float)));
	const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
	const dim3 gridSize(16, 16,1);
	gpu_projection<<<gridSize,blockSize>>>(d_output,xmin,xmax,ymin,ymax);
	gpuErrchk(hipPeekAtLastError());
	hipDeviceSynchronize();
	gpuErrchk(hipMemcpy(outgg,d_output,DetectX*DetectZ*frameN*sizeof(float),hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_output));
	hipUnbindTexture(img_tex);
	gpuErrchk(hipFreeArray(cuArray_a));
	//gpuErrchk(hipFree(d_ggin));
	end=clock();
	time=(end-start)/CLK_TCK;
	printf("Projection Succeed!time=%.5f\n",time);
}
extern "C"
void gpu_Artprojection( float *img,float* outgg,int xmin,int xmax,int ymin,int ymax,int depth)
{
	printf("Start Projection calculating...\n");
	clock_t start,end;
	float time;
	start=clock();
	float* d_output=NULL;
	float* d_imgin=NULL;
	double* z0=NULL;double*zx=NULL;double* zy=NULL;double *zz=NULL;
	long * J=NULL;
	double* L=NULL;
	//caculate_WD();
	theta_init();
	gpuErrchk(hipMalloc((void**)&z0,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMemset((void*)z0,0,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&zx,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMemset((void*)zx,0,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&zy,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMemset((void*)zy,0,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&zz,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMemset((void*)zz,0,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&J,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMemset((void*)J,0,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&L,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMemset((void*)L,0,(imageN*3+1)*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&d_output,DetectX*DetectZ*frameN*sizeof(float)));
	gpuErrchk(hipMemset((void*)d_output,0,DetectX*DetectZ*frameN*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_imgin,imageX*imageY*(ztop-zbot)*sizeof(float)));
	gpuErrchk(hipMemcpy(d_imgin,img,imageX*imageY*(ztop-zbot)*sizeof(float),hipMemcpyHostToDevice));
	gpuErrchk(hipMemset((void*)d_output,0,DetectX*DetectZ*frameN*sizeof(float)));
	const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
	const dim3 gridSize((DetectX+blockSize.x-1) / blockSize.x, (DetectZ+blockSize.y-1) / blockSize.y,1);
	//gpu_projection<<<gridSize,blockSize>>>(d_output,xmin,xmax,ymin,ymax);
	get_projection<<<gridSize,blockSize>>>(d_imgin,d_output,z0,zx,zy,zz,J,L);
	gpuErrchk(hipPeekAtLastError());
	hipDeviceSynchronize();
	gpuErrchk(hipMemcpy(outgg,d_output,DetectX*DetectZ*frameN*sizeof(float),hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_output));
	gpuErrchk(hipFree(d_imgin));
	gpuErrchk(hipFree(z0));
	gpuErrchk(hipFree(zx));
	gpuErrchk(hipFree(zy));
	gpuErrchk(hipFree(zz));
	gpuErrchk(hipFree(J));
	gpuErrchk(hipFree(L));
	end=clock();
	time=(end-start)/CLK_TCK;
	printf("Projection Succeed!time=%.5f\n",time);
}

extern "C"
void gpu_fdkbackprj(float *prj,float *img,int width,int height,int imageW,int imageH,int bottom,int top)
{
	printf("Start BackProjection calculating...\n");
	clock_t start,end;
	float time;
	start=clock();
	theta_init();
	int k;
	unsigned int i;
	const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
	const dim3 gridSize((imageW+blockSize.x-1) / blockSize.x, (imageH+blockSize.y-1) / blockSize.y,1);
	float *d_outimg=NULL;
	float *d_gg=NULL;
	gpuErrchk(hipMalloc((void**)&d_outimg,imageH*imageW*(top-bottom)*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_gg,width*height*frameN*sizeof(float)));
	gpuErrchk(hipMemset((void*)d_outimg,0,imageH*imageW*(top-bottom)*sizeof(float)));
	gpuErrchk(hipMemcpy(d_gg,prj,height*width*frameN*sizeof(float),hipMemcpyHostToDevice));
	/*hipArray* cuArray=allocateProjectionArray(width,height,frameN);
	if(!transferProjectionsToArray( prj, cuArray,  width, height,frameN))
		return;
	else
		if(!bindProjDataTexture(cuArray))return;	*/
	gpu_fdk<<<gridSize, blockSize>>>(d_gg,d_outimg,imageW,imageH,zbot,ztop);
	gpuErrchk(hipPeekAtLastError());
	hipDeviceSynchronize();
	
	gpuErrchk(hipMemcpy(img,d_outimg,imageW*imageH*(top-bottom)*sizeof(float),hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_outimg));

	gpuErrchk(hipFree(d_gg));
	end=clock();
	time=(end-start)/CLK_TCK;
	printf("BacProjection Succeed!time=%.5f\n",time);
}
extern "C"
void gpu_backprj(float *prj,float *img,int width,int height,int imageW,int imageH,int bottom,int top)
{
	printf("Start BackProjection calculating...\n");
	clock_t start,end;
	float time;
	start=clock();
	theta_init();
	int k;
	unsigned int i;
	const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
	const dim3 gridSize((imageW+blockSize.x-1) / blockSize.x, (imageH+blockSize.y-1) / blockSize.y,1);
	float *d_outimg=NULL;
	float *d_gg=NULL;
	gpuErrchk(hipMalloc((void**)&d_outimg,imageH*imageW*(top-bottom)*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_gg,width*height*frameN*sizeof(float)));
	gpuErrchk(hipMemset((void*)d_outimg,0,imageH*imageW*(top-bottom)*sizeof(float)));
	gpuErrchk(hipMemcpy(d_gg,prj,height*width*frameN*sizeof(float),hipMemcpyHostToDevice));
	/*hipArray* cuArray=allocateProjectionArray(width,height,frameN);
	if(!transferProjectionsToArray( prj, cuArray,  width, height,frameN))
		return;
	else
		if(!bindProjDataTexture(cuArray))return;	*/
	gpu_backProjection<<<gridSize, blockSize>>>(d_gg,d_outimg,imageW,imageH,zbot,ztop);
	gpuErrchk(hipPeekAtLastError());
	hipDeviceSynchronize();
	
	gpuErrchk(hipMemcpy(img,d_outimg,imageW*imageH*(top-bottom)*sizeof(float),hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_outimg));

	gpuErrchk(hipFree(d_gg));
	end=clock();
	time=(end-start)/CLK_TCK;
	printf("BacProjection Succeed!time=%.5f\n",time);
}
////////////////////////////////////////////////////////////////////////////////

void gpu_release()
{
	//gpuErrchk(hipFree(d_gg));
}

void caculate_WD()
{  
	float* WDO1=new float[frameN];
	float M[N0],N[N0],Dpq[N0][N0],up[N0],thitap[N0],wq[N0],aver_dp[N0],aver_up[N0],aver_thitap[N0],Dp[N0];
	float tempt0,tempt1,sum0,sum1;
	int i,j,k,p,q;
	for(i=0;i<N0-1;i++)
	{
		M[i] = (i+1)*360.0/frameN;   //保存未进行加权距离正交计算的角度
		N[i] = 0;     //保存进行加权距离正交计算后的角度
	}
	int L,Q;
	Q=1;
	for(L=N0-1;L>0;L--)
	{
		for(p=0;p<L;p++)
			for(q=0;q<Q;q++)
			{ tempt0 = abs(M[p] - N[q]);
		Dpq[p][q] =  tempt0 < 90 - tempt0 ? tempt0 : 90-tempt0;
		}  //计算投影p、q之间的 距离


		for(q=0;q<Q;q++)
			wq[q] = (q+1)*1.0/Q ; //

		sum0 = 0;
		for(q=0;q<Q;q++)
			sum0 += wq[q]; //

		for(p=0;p<L;p++)
		{ sum1 = 0;
		for(q=0;q<Q;q++)
			sum1 += wq[q]*(45-Dpq[p][q]) ;
		up[p] = sum1*1.0/sum0;
		}    //

		for(p=0;p<L;p++)
		{   sum1 =0;
		for(q=0;q<Q;q++)
			sum1 += Dpq[p][q];
		aver_dp[p] = sum1/Q ;
		}//

		for(p=0;p<L;p++)
		{  sum1 =0;
		for(q=0;q<Q;q++)
			sum1 += wq[q]*(Dpq[p][q] - aver_dp[p])*(Dpq[p][q] - aver_dp[p]);
		thitap[p] = sqrt(sum1/sum0);
		}
		//

		float min_up,max_up,min_thitap,max_thitap;
		min_up = up[0];
		max_up = up[0];
		min_thitap = thitap[0];
		max_thitap = thitap[0];
		for(p=1;p<L;p++)
		{
			if(min_up > up[p] ) min_up = up[p];
			if(max_up < up[p] ) max_up = up[p];
			if(min_thitap > thitap[p] ) min_thitap = thitap[p] ;
			if(max_thitap < thitap[p] ) max_thitap = thitap[p] ;

		}		


		for(p=0;p<L;p++)
		{
			aver_up[p] = (up[p] - min_up)/(max_up -min_up);
			aver_thitap[p] = (thitap[p] - min_thitap)/(max_thitap - min_thitap);
			Dp[p] = aver_up[p]*aver_up[p] + 0.5*aver_thitap[p]*aver_thitap[p];
		}

		float min_D;
		min_D = Dp[0];
		k=0;
		for(p=1;p<L;p++)
		{
			if(min_D>Dp[p])
			{
				min_D = Dp[p];
				k=p;
			}

		}
		N[Q] = M[k];
		M[k] = M[L-1];

		Q++ ;

	}

	N[N0-1] = M[0];
	for(k=0;k<N0;k++)
	{
		WDO1[k*4] = N[k]*frameN/360;
		WDO1[k*4+1] =( N[k] +90)*frameN/360;
		WDO1[k*4+2] =(N[k]+ 180)*frameN/360;
		WDO1[k*4 + 3] = (N[k] + 270)*frameN/360;
	}
gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(gc_WDO), WDO1, frameN*sizeof(float), 0, hipMemcpyHostToDevice));
}